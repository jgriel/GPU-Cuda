#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"

#define N_THREADS 1024
#define N_BLOCKS 16

/*** GPU functions ***/
__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);
}

__global__ void random_walk_kernel(float *map, int rows, int cols, int* bx, int* by,
                                   int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement random walk!
}

__global__ void local_max_kernel(float *map, int rows, int cols, int* bx, int* by,
                                 int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max!
}

__global__ void local_max_restart_kernel(float *map, int rows, int cols, int* bx,
                                         int* by, int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max with restarts!
}

/*** CPU functions ***/
hiprandState* init_rand() {
  hiprandState *d_state;
  hipMalloc(&d_state, N_BLOCKS * N_THREADS * sizeof(hiprandState));
  init_rand_kernel<<<N_BLOCKS, N_THREADS>>>(d_state);
  return d_state;
}


float random_walk(float* map, int rows, int cols, int steps) {
  hiprandState* d_state = init_rand();
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;

  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU

  random_walk_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  // After kernel call:
  // Need to copy data back to CPU and find max value

  float max_val = 0;

  // Finally: free used GPU and CPU memory


  return max_val;
}

// Work on these after finishing random walk
float local_max(float* map, int rows, int cols, int steps);
float local_max_restart(float* map, int rows, int cols, int steps);


int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s <map_file> \n", argv[0]);
    return 1;
  }

  float *map;
  int rows, cols;
  read_bin(argv[1], &map, &rows, &cols);

  printf("%d %d\n", rows, cols);

  // As a starting point, try to get it working with a single steps value
  int steps = 10;
  float max_val = random_walk(map, rows, cols, steps);
  printf("Random walk max value: %f\n", max_val);

  return 0;
}
